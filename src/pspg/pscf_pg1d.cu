#include <pspg/System.h>

int main(int argc, char **argv)
{
   
   Pscf::Pspg::System<1> system;

   // Process command line options
   system.setOptions(argc, argv);

   // Read parameters from default parameter file
   system.readParam();

   // Read command script to run system
   system.readCommands();
   //hipDeviceReset();
   return 0;
}